#include "hip/hip_runtime.h"
extern "C"
__global__ void __kernel_name(float *arg_dt, int *arg_rest, float *block_parameters, float *idata, float *sdata, float *odata)
{
    // The kernel needs to allocate all the shared memory of an SM
    // for each block so that only 1 block runs on an SM

    // Need to be able to determine what the shared memory size is.
    extern __shared__ float local_sdata[];

    // Array size is dependent on nodes * (sv + trace var) * record_time_steps
    unsigned int n_idx = threadIdx.x;
    unsigned int n_mem_idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int num_blocks = gridDim.x;
    const unsigned int num_vars = __num_vars;
    const unsigned int num_track_vars = __num_track_vars;
    const unsigned int num_shared_vars = __num_shared_vars;

    // runtime settable
    const int num_nodes = arg_rest[0];
    const int num_time_steps = arg_rest[1];
    const int iterations_per_record = arg_rest[2];
    const int num_parameters = arg_rest[3];
    const float dt = arg_dt[0];
    unsigned int parameter_offset = blockIdx.x*num_parameters;

    __set_block_parameters;

    // Shared State Variable Indexes
    __shared_sv_indexes;
    
    // State Variable Declarations
    __state_variable_declarations;

    // shared algebra declarations
    __shared_algebra_declarations;
    
    // Initial Conditions
    __initial_conditions;

    // Compute Loop
    unsigned int ts_idx;
    unsigned int record_cycler = 1;
    unsigned int record_ts_idx;
    if (n_idx < num_nodes)
	for (ts_idx = 1; ts_idx < num_time_steps; ++ts_idx)
	    {
		/// Read Shared Data
		// Note: We put only set from the values we care about
		//__threadfence_block();
		__shared_sv_read;

		// compute
		__compute;

		// Write Shared Data
		__shared_sv_write;
		    
		// Record Data
		if (record_cycler == iterations_per_record)
		    {
			record_ts_idx = ts_idx/iterations_per_record;
			__write_data;
			record_cycler = 1;
		    }
		else
		    {
			++record_cycler;
		    }
	    }
}
